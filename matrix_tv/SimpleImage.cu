#include <stdlib.h>
#include <iostream>
#include <hip/hip_runtime.h>
#include "SimpleImage.h"

using std::cerr;
void SimpleImage::allocate_2D_image(size_t _width,size_t _height,SimpleImage::T *h_img, size_t _depth)
{
    height=_height;
    width=_width;
    size_t bwidth=width*sizeof(SimpleImage::T);
    depth=_depth;
    channels=1;
    gpu_data=true;
    cerr<<"height= "<<height<<", width= "<<width<<"\n";
    hipError_t err;

    hipExtent extent = make_hipExtent(bwidth, height,depth);
    hipPitchedPtr dest_ptr;
    err=hipMalloc3D(&dest_ptr,extent);
    if (!h_img)
    {
        this->data=(T*)dest_ptr.ptr;
        err=hipMemset3D (dest_ptr,0,extent);
    }
    else
    {
        hipPitchedPtr src_ptr=make_hipPitchedPtr((void *)h_img,bwidth,width,height);
        size_t pitch=dest_ptr.pitch;
        if (err!=hipSuccess)
        {
            cerr<<"Error on allocating pitched memory: "<<hipGetErrorString(err)<<"\n";
        }
        hipMemcpy3DParms myParms = {0};
        myParms.extent=extent;
        myParms.srcPtr=src_ptr;
        myParms.dstPtr=dest_ptr;
        cerr<<"hipMalloc3D(2): allocated at "<<dest_ptr.ptr<<"\n";
        this->data=(float *)dest_ptr.ptr;
        this->pitch=dest_ptr.pitch;
        myParms.kind=hipMemcpyHostToDevice;

        cerr<<"dpitch = "<<pitch<<", spitch="<<bwidth<<", width="<<bwidth<<", pitch="<<pitch<<"\n";
        cerr<<"from: "<<h_img<<", to: "<<this->data<<"\n";
        cerr<<"hipMemcpyHostToDevice\n";
        cerr<<"h_img.pitch = "<<pitch<<"\n";
        err=hipMemcpy3D(&myParms);
        cerr<<"..\n";
    }
    if (err!=hipSuccess)
    {
        cerr<<"Error on copying to pitched memory: "<<hipGetErrorString(err)<<"\n";
    }

    cerr<<"SimpleImage.allocate_2D_image: data="<<this->data<<"\n";
}




void SimpleImage::allocate_2D_image(const SimpleImage &h_img)
{
    height=h_img.height;
    width=h_img.width;
    depth=h_img.depth;
    channels=1;
    gpu_data=true;
    cerr<<"height= "<<height<<", width= "<<width<<"\n";
    //hipError_t err=hipMallocPitch(&this->data,&pitch,width*sizeof(SimpleImage::T),height);
    hipPitchedPtr src_ptr=make_hipPitchedPtr((void *)h_img.data,h_img.pitch,h_img.width,h_img.height);

    size_t bwidth=width*sizeof(SimpleImage::T);
    hipExtent extent = make_hipExtent(bwidth, height,depth);
    hipPitchedPtr dest_ptr;
    hipError_t err=hipMalloc3D(&dest_ptr,extent);

    if (err!=hipSuccess)
    {
        cerr<<"Error on allocating pitched memory: "<<hipGetErrorString(err)<<"\n";
    }
    hipMemcpy3DParms myParms = {0};

    myParms.srcPtr=src_ptr;
    myParms.dstPtr=dest_ptr;
    myParms.extent=extent;
    cerr<<"hipMalloc3D(2): allocated at "<<dest_ptr.ptr<<"\n";
    this->data=(float *)dest_ptr.ptr;
    this->pitch=dest_ptr.pitch;

    cerr<<"dpitch = "<<pitch<<", spitch="<<bwidth<<", width="<<bwidth<<", pitch="<<pitch<<"\n";
    cerr<<"from: "<<h_img.data<<", to: "<<data<<"\n";
    if (h_img.gpu_data)
    {
        cerr<<"hipMemcpyDeviceToDevice\n";
        myParms.kind=hipMemcpyDeviceToDevice;
        err=hipMemcpy3D(&myParms);
    }
    else
    {
        cerr<<"hipMemcpyHostToDevice\n";
        cerr<<"h_img.pitch = "<<h_img.pitch<<"\n";
        myParms.kind=hipMemcpyHostToDevice;
        err=hipMemcpy3D(&myParms);
        //   err=hipMemcpy2D(this->data, pitch, h_img.data, h_img.pitch, bwidth, height, hipMemcpyHostToDevice);
    }

    //err=hipMemcpy2D(data, pitch, h_img.data, bwidth, bwidth, height, hipMemcpyHostToDevice);
//err=hipMemcpy2D(data, pitch, h_img, sizeof(SimpleImage::T), 1, 1, hipMemcpyHostToDevice);

    if (err!=hipSuccess)
    {
        cerr<<"Error on copying to pitched memory: "<<hipGetErrorString(err)<<"\n";
    }

    cerr<<"SimpleImage.allocate_2D_image: data="<<this->data<<"\n";
//	height=_height;width=_width;depth=1;channels=1;gpu_data=true;
//	cerr<<"(2)height= "<<height<<", width= "<<width<<"\n";
//	hipError_t err=hipMallocPitch(&this->data,&pitch,width*sizeof(SimpleImage::T),height);
//if (err!=hipSuccess){
//cerr<<"Error on allocating pitched memory(2): "<<hipGetErrorString(err)<<"\n";
//}
////size_t bwidth=width*sizeof(SimpleImage::T);
//size_t bwidth=width*sizeof(SimpleImage::T);
//cerr<<"dpitch = "<<pitch<<", spitch="<<bwidth<<", width="<<bwidth<<"\n";
//cerr<<"from: "<<h_img.data<<", to: "<<data<<"\n";
//
//err=hipMemcpy2D(data, pitch, h_img.data, bwidth, bwidth, height, hipMemcpyHostToDevice);
////cerr<<"(2)width="<<width<<"dpitch = "<<dpitch<<", width = "<<width<<", bwidth="<<bwidth<<"\n";
////err=hipMemcpy2D(data, dpitch, h_img.data, bwidth, bwidth, height, hipMemcpyHostToDevice);
//if (err!=hipSuccess){
//cerr<<"Error on copying to pitched memory(2): "<<hipGetErrorString(err)<<"\n";
//}
//
}
void SimpleImage::copy_to_host(SimpleImage::T *h_img)
{
    hipError_t err;
    if (gpu_data)
    {
        cerr<<"Copying GPU data to host..\n";
        cerr<<"height: "<<height<<", width: "<<width<<", depth: "<<depth<<", channels: "<<channels<<"\n";
        cerr<<"from: "<<data<<", to: "<<h_img<<"\n";
//err=hipMemcpy( (void*)h_img,(void*)data, pitch*height*depth*channels,hipMemcpyDeviceToHost);
        err=hipMemcpy( (void*)h_img,(void*)data, pitch*height*depth*channels,hipMemcpyDeviceToHost);
//err=hipMemcpy2D( (void*)h_img,(void*)pitch, data*height*depth*channels,hipMemcpyDeviceToHost);
    }
    else
    {
        cerr<<"Copying CPU data to host..\n";
        err=hipMemcpy( (void*)h_img,(void*)data, pitch*height*depth*channels,hipMemcpyHostToHost);

    }
    if (err!=hipSuccess)
    {
        cerr<<"Error on copying memory to host: "<<hipGetErrorString(err)<<"\n";
    }
}

void SimpleImage::copy_to_host(SimpleImage& h_img)
{
    hipError_t err;
    if (gpu_data && !h_img.gpu_data)
    {
        cerr<<"Copying GPU data to host (2)..\n";
        size_t bpitch=pitch;//*sizeof(SimpleImage::T);
        size_t dpitch=h_img.pitch;
        cerr<<"GPU pitch: "<<bpitch<<", host pitch: "<<dpitch<<"\n";
        cerr<<"height: "<<height<<", width: "<<width<<", depth: "<<depth<<", channels: "<<channels<<"\n";
        cerr<<"from: "<<data<<", to: "<<h_img.data<<"\n";
        //err=hipMemcpy2D( (void *)h_img.data, dpitch, (void*)this->data,bpitch, width*sizeof(T),height,hipMemcpyDeviceToHost);
        size_t bwidth=width*sizeof(T);
        hipPitchedPtr src_ptr=make_hipPitchedPtr((void *)data,pitch,width,height);
        hipPitchedPtr dest_ptr=make_hipPitchedPtr((void *)h_img.data,h_img.pitch,h_img.width,h_img.height);

        hipExtent extent = make_hipExtent(bwidth, height,depth);
        hipMemcpy3DParms myParms = {0};

        myParms.srcPtr=src_ptr;
        myParms.dstPtr=dest_ptr;
        myParms.extent=extent;
        myParms.kind=hipMemcpyDeviceToHost;
        err=hipMemcpy3D(&myParms);

        if (err!=hipSuccess)
        {
            cerr<<"Error on copying memory to host: "<<hipGetErrorString(err)<<"\n";
        }
//err=hipMemcpy( (void*)h_img.data,(void*)data, 1,hipMemcpyDeviceToHost);
    }
    else
    {
//cerr<<"Copying CPU data to host..\n";
        cerr<<"TBD\n";
//err=hipMemcpy2D( h_img.data, h_img.pitch*sizeof(SimpleImage::T), data,bpitch, width,height,hipMemcpyDeviceToHost);
    }
}

SimpleImage::SimpleImage(unsigned int _width,unsigned int _height,unsigned int _depth,SimpleImage::T*_data, bool _gpu_data):width(_width),height(_height),depth(_depth),gpu_data(_gpu_data)
{
    pitch=_width*sizeof(T); // todo - replace in a different constructor..
    data=_data;
    //channels=0;
    //ctor
}

SimpleImage::~SimpleImage()
{
}
void SimpleImage::dealloc()
{
    if (data!=0)
    {

        if (gpu_data)
        {
            cerr<<"Freeing gpu data at "<<data<<"\n";
            hipFree(data);
        }
        else
        {
            cerr<<"Freeing cpu data at"<<data<<"\n";
            delete[] data;
        }
        data=0;
    }
    //dtor
}



